#include "hip/hip_runtime.h"
#include "quant_cuda.h"
#include "quant_kernel.h"
#include <ATen/ATen.h>
#include <climits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <tuple>

using namespace at;

Tensor get_max_entry(Tensor a, int dim) {
  Tensor max_entry;
  if (dim == -1) {
    max_entry = at::max(at::abs(a)).expand_as(a).contiguous();
  } else if (dim == 0) {
    Tensor input_view = a.view({a.size(0), -1});
    max_entry = std::get<0>(input_view.abs().max(1, true))
                    .expand_as(input_view)
                    .view_as(a)
                    .contiguous();
  } else {
    Tensor input_transpose = a.transpose(0, dim);
    Tensor input_view =
        input_transpose.contiguous().view({input_transpose.size(0), -1});
    Tensor max_transpose = std::get<0>(input_view.abs().max(1, true))
                               .expand_as(input_view)
                               .view_as(input_transpose);
    max_entry = max_transpose.transpose(dim, 0).contiguous();
  }
  return max_entry;
}

Tensor block_quantize_stochastic_cuda(Tensor a, int wl, int dim) {
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_nearest_cuda(Tensor a, int wl, int dim) {
  auto o = at::zeros_like(a);
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_stochastic_cuda(Tensor a, int wl) {
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_nearest_cuda(Tensor a, int wl) {
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor float_quantize_stochastic_cuda(Tensor a, int man_bits, int exp_bits,
                                      bool subnormals, bool saturate) {
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      man_bits, exp_bits, subnormals, saturate);
  return o;
}

Tensor float_quantize_nearest_cuda(Tensor a, int man_bits, int exp_bits,
                                   bool subnormals, bool saturate) {
  // use external random number right now
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits,
      subnormals, saturate);
  return o;
}

void fixed_min_max(int wl, int fl, bool symmetric, float *t_min, float *t_max) {
  int sigma = -fl;
  *t_min = -ldexp(1.0, wl - fl - 1);
  *t_max = -*t_min - ldexp(1.0, sigma);
  if (symmetric)
    *t_min = *t_min + ldexp(1.0, sigma);
}

Tensor fixed_point_quantize_stochastic_cuda(Tensor a, int wl, int fl,
                                            bool use_clamp, bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, sigma, use_clamp, t_min, t_max);
  return o;
}

Tensor fixed_point_quantize_nearest_cuda(Tensor a, int wl, int fl,
                                         bool use_clamp, bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, sigma, use_clamp, t_min,
      t_max);
  return o;
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_stochastic_mask_cuda(Tensor a, int wl, int fl,
                                          bool symmetric) {
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_probs = rand_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      m.data_ptr<uint8_t>(), size, sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_nearest_mask_cuda(Tensor a, int wl, int fl,
                                       bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), m.data_ptr<uint8_t>(), size,
      sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

__global__ void init(unsigned int seed, hiprandState_t *state) {
  hiprand_init(seed, blockIdx.x * blockIdx.y, 0,
              &state[blockIdx.x * blockIdx.y]);
}

void float_quantize_nearest_gemm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                      int N, int K, int man_add, int exp_add,
                                      int man_mul, int exp_mul, bool subnormals,
                                      bool saturate) {

  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  gemm_fp_nearest<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(), c.data<float>(), M, K, N, man_add,
      exp_add, man_mul, exp_mul, subnormals, saturate);

  return;
}

void float_quantize_nearest_gemm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                          int N, int K, int man_fma,
                                          int exp_fma, bool subnormals,
                                          bool saturate) {

  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  gemm_fp_fma_nearest<<<blocks, threads>>>(a.data<float>(), b.data<float>(),
                                           c.data<float>(), M, K, N, man_fma,
                                           exp_fma, subnormals, saturate);

  return;
}

void float_quantize_stochastic_gemm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                         int N, int K, int man_add, int exp_add,
                                         int man_mul, int exp_mul,
                                         bool subnormals, bool saturate) {

  // auto rand_ints = randint(INT_MAX, {(M + 8 - M % 8) * (N + 8 - N % 8) * (K +
  // 8 - K % 8) * 2},
  //                    device(kCUDA).dtype(kInt));
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  hiprandState_t *state;
  hipMalloc((void **)&state, blocks.x * blocks.y * sizeof(hiprandState_t));
  init<<<blocks, 1>>>(time(0), state);
  gemm_fp_stochastic<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(), c.data<float>(),
      state, // rand_ints.data<int>(),
      M, K, N, man_add, exp_add, man_mul, exp_mul, subnormals, saturate);
  hipFree(state);
  return;
}

void float_quantize_stochastic_gemm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                             int M, int N, int K, int man_fma,
                                             int exp_fma, bool subnormals,
                                             bool saturate) {

  auto rand_ints =
      randint(INT_MAX, {(M + 8 - M % 8) * (N + 8 - N % 8) * (K + 8 - K % 8)},
              device(kCUDA).dtype(kInt));
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  hiprandState_t *state;
  hipMalloc((void **)&state, blocks.x * blocks.y * sizeof(hiprandState_t));
  init<<<blocks, 1>>>(time(0), state);
  gemm_fp_fma_stochastic<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(), c.data<float>(),
      state, // rand_ints.data<int>(),
      M, K, N, man_fma, exp_fma, subnormals, saturate);
  hipFree(state);
  return;
}

void fixed_point_quantize_nearest_gemm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                            int N, int K, int wl_add,
                                            int fl_add, int wl_mul, int fl_mul,
                                            bool symmetric) {
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  gemm_fxp_nearest<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(), c.data<float>(), M, K, N, sigma_add,
      t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_nearest_gemm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                int M, int N, int K, int wl_fma,
                                                int fl_fma, bool symmetric) {
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  gemm_fxp_fma_nearest<<<blocks, threads>>>(a.data<float>(), b.data<float>(),
                                            c.data<float>(), M, K, N, sigma_fma,
                                            t_min_fma, t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_gemm_cuda(Tensor a, Tensor b, Tensor c,
                                               int M, int N, int K, int wl_add,
                                               int fl_add, int wl_mul,
                                               int fl_mul, bool symmetric) {
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  // auto rand_probs = at::rand({(M + 8 - M % 8), (N + 8 - N % 8), (K + 8 - K %
  // 8) * 2},
  //                     device(kCUDA).dtype(kFloat));
  hiprandState_t *state;
  hipMalloc((void **)&state, blocks.x * blocks.y * sizeof(hiprandState_t));
  // TODO: change this to a fixed seed?!
  init<<<blocks, 1>>>(time(0), state);
  gemm_fxp_stochastic<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(),
      c.data<float>(), // rand_probs.data<float>(), M, K, N,
      state, M, K, N, sigma_add, t_min_add, t_max_add, sigma_mul, t_min_mul,
      t_max_mul);
  hipFree(state);
  return;
}

void fixed_point_quantize_stochastic_gemm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                   int M, int N, int K,
                                                   int wl_fma, int fl_fma,
                                                   bool symmetric) {
  dim3 threads(8, 8);
  dim3 blocks((N + 8 - N % 8) / 8, (M + 8 - M % 8) / 8);
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  // auto rand_probs = at::rand({(M + 8 - M % 8) * (N + 8 - N % 8) * (K + 8 - K
  // % 8)},
  //                     device(kCUDA).dtype(kFloat));
  hiprandState_t *state;
  hipMalloc((void **)&state, blocks.x * blocks.y * sizeof(hiprandState_t));
  init<<<blocks, 1>>>(time(0), state);
  gemm_fxp_fma_stochastic<<<blocks, threads>>>(
      a.data<float>(), b.data<float>(),
      c.data<float>(), // rand_probs.data<float>(), M, K, N,
      state, M, K, N, sigma_fma, t_min_fma, t_max_fma);
  hipFree(state);
  return;
}